#include "hip/hip_runtime.h"
// runSim.cu

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

// Executes the A1 operator optimized
__global__ void A1_kernel(double* r, double* v, double dt) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    r[id] += v[id] * dt;
}

// Executes the A2 operator
__global__ void A2_kernel(double *r, double *v, double *m, double dt, double *varr, double *status, int numParticles) {
	size_t id = blockIdx.x * blockDim.x + threadIdx.x + 1;
	double invdist;
	double dirvec[3];

	if (id < numParticles) {
		dirvec[0] = r[0] - r[3*id];
		dirvec[1] = r[1] - r[3*id+1];
		dirvec[2] = r[2] - r[3*id+2];

		// Distance between particle 0 and i
		invdist = dt * rnorm3d(dirvec[0], dirvec[1], dirvec[2])*\
					   rnorm3d(dirvec[0], dirvec[1], dirvec[2])*\
					   rnorm3d(dirvec[0], dirvec[1], dirvec[2]);
		
		if (status[id] == 0) {
        	v[3*id]   += 0;
        	v[3*id+1] += 0;
        	v[3*id+2] += 0;

        	varr[id]                = 0;
        	varr[numParticles+id]   = 0;
        	varr[2*numParticles+id] = 0;
		}		
		else {	
			// Update velocities of particles 1 through N-1
			v[3*id]   += m[0] * invdist * dirvec[0];
			v[3*id+1] += m[0] * invdist * dirvec[1];
			v[3*id+2] += m[0] * invdist * dirvec[2];

			varr[id]                = -m[id] * invdist * dirvec[0];
			varr[numParticles+id]   = -m[id] * invdist * dirvec[1];
			varr[2*numParticles+id] = -m[id] * invdist * dirvec[2];
		}

        varr[0]              = v[0];
        varr[numParticles]   = v[1];
    	varr[2*numParticles] = v[2];
	}
}

// Execute the B operator when only embryo and other particles interact
__global__ void B_kernel(double *r, double *v, double *m, double *varr, double dt, int numParticles, double *status, double eps) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x + 2;
    double dirvec[3];
    double invdist;

    if (id < numParticles) {
        dirvec[0] = r[3]   - r[3*id];
        dirvec[1] = r[3+1] - r[3*id+1];
        dirvec[2] = r[3+2] - r[3*id+2];

        invdist = status[id] * dt * rsqrt((dirvec[0]*dirvec[0] + dirvec[1]*dirvec[1] + dirvec[2]*dirvec[2] + eps*eps)*\
					      	 			  (dirvec[0]*dirvec[0] + dirvec[1]*dirvec[1] + dirvec[2]*dirvec[2] + eps*eps)*\
  					     	 			  (dirvec[0]*dirvec[0] + dirvec[1]*dirvec[1] + dirvec[2]*dirvec[2] + eps*eps));

		// update id'th satelitesimal 
        v[3*id]   += m[1] * invdist * dirvec[0];
        v[3*id+1] += m[1] * invdist * dirvec[1];
        v[3*id+2] += m[1] * invdist * dirvec[2];

        // update embryo
        // Store forces on embryo for reduction
        varr[0]                = v[3];
		varr[numParticles-1]   =    0;
        varr[numParticles]     = v[4];
		varr[2*numParticles-1] =    0;
        varr[2*numParticles]   = v[5];
		varr[3*numParticles-1] =    0;

        varr[id-1]                = -m[id] * invdist * dirvec[0];
        varr[numParticles+id-1]   = -m[id] * invdist * dirvec[1];
        varr[2*numParticles+id-1] = -m[id] * invdist * dirvec[2];
	}
}

__global__ void mergeEject(double *r, double *status, int numParticles, double rH) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x + 2;
    double dist;

    if (id < numParticles) {
        dist = norm3d(r[0]-r[3*id], r[1]-r[3*id+1], r[2]-r[3*id+2]);

        if (dist < 0.03*rH && status[id] != 0) 
            status[id] = 2;
    	else if (dist > rH && status[id] != 0)
			status[id] = 3;  // so that momentum conservation doesn't include ejected particles
							 // will be set to 0 in the consMomentum function
	}
}

__global__ void consMomentum(double *v, double *m, double *status, int numParticles, double *rSatellites) {
	for (int id = 2; id < numParticles; id++) {
		if (status[id] == 2) {
			status[id] 		= 0;
			// use conservation of momentum to update central velocity
    		v[0]       		= 1./(m[0] + m[id]) * (m[0]*v[0] + m[id]*v[3*id]);
    		v[1]       		= 1./(m[0] + m[id]) * (m[0]*v[1] + m[id]*v[3*id+1]);
    		v[2]       		= 1./(m[0] + m[id]) * (m[0]*v[2] + m[id]*v[3*id+2]);
    		// conservation of mass
    		m[0] 		   += m[id];
		}
		else if (status[id] == 4) {
			status[id] 	       = 0;
			rSatellites[0]     = cbrt((m[1]+m[2])/m[2])*rSatellites[1];
			// use conservation of momentum to update velocity
            v[3]   			   = 1./(m[1] + m[id]) * (m[1]*v[3] + m[id]*v[3*id]);
            v[4] 			   = 1./(m[1] + m[id]) * (m[1]*v[4] + m[id]*v[3*id+1]);
            v[5] 			   = 1./(m[1] + m[id]) * (m[1]*v[5] + m[id]*v[3*id+2]);
            // conservation of mass
            m[1]    		  += m[id];
		}
		else if (status[id] == 3)
			status[id] = 0;
		else
			continue;
	}
}

__global__ void statusUpdate(double *r, double *v, double *m, double *status, int numParticles) {
	size_t id = blockIdx.x * blockDim.x + threadIdx.x;

    m[id/3] *= status[id/3];
   	r[id] 	*= status[id/3];
    v[id] 	*= status[id/3];
}

// Function to find 
// cross product of two vector array. 
__device__ void crossProduct(double *vect_A, double *vect_B, double *cross_P) { 
    cross_P[0] = vect_A[1] * vect_B[2] - vect_A[2] * vect_B[1]; 
    cross_P[1] = vect_A[2] * vect_B[0] - vect_A[0] * vect_B[2]; 
    cross_P[2] = vect_A[0] * vect_B[1] - vect_A[1] * vect_B[0]; 
} 

__global__ void collision(double* r, double* v, double* status, double* rSatellites, int numParticles, double dt) {
	size_t id = blockIdx.x * blockDim.x + threadIdx.x + 2;

    double rTemp[3]; 
    double vTemp[3];
    double crossP[3];
    double vecA[3];
    double vecB[3];
    double t;
    double dist;
    double d1;
    double d2;

	if (id < numParticles) {
    	// go to rest frame of embryo
    	vTemp[0] = v[3*id]   - v[3];
    	vTemp[1] = v[3*id+1] - v[4];
    	vTemp[2] = v[3*id+2] - v[5];

    	// evolve satelitesimal
    	rTemp[0] = r[3*id]   + vTemp[0] * dt/4.0;
    	rTemp[1] = r[3*id+1] + vTemp[1] * dt/4.0;
    	rTemp[2] = r[3*id+2] + vTemp[2] * dt/4.0;

    	// the equation ((r-r[1]) * (rTemp-r)) / |rTemp-r|^2 where r[1] is the embryo's
    	// position in its rest frame, r is the satelitesimal's original position and rTemp is the
    	// satelitesimal's updated position in the rest frame. * indicates a dot product in this case
    	// this is the time that minimizes the distance function from a line segment to a point
    	t = -1*((r[3*id]-r[3])      *(rTemp[0]-r[3*id])    +\
         	  	(r[3*id+1]-r[4])    *(rTemp[1]-r[3*id+1])  +\
         	  	(r[3*id+2]-r[5])    *(rTemp[2]-r[3*id+2])) /\
        	   ((rTemp[0]-r[3*id])  *(rTemp[0]-r[3*id])    +\
         	  	(rTemp[1]-r[3*id+1])*(rTemp[1]-r[3*id+1])  +\
         	  	(rTemp[2]-r[3*id+2])*(rTemp[2]-r[3*id+2]));

    	if (0 < t < 1) {
    		// the equation |(r[1]-r) x (r[1]-rTemp)|/|rTemp-r| where r[1] is the embryo's position
    		// in its rest frame, r is the satelitesimal's original position and rTemp is the
    		// satelitesimal's updated position in the rest frame
    		// if t is in this range, then the point in within line segment
 			vecA[0] = r[3]-r[3*id],  vecA[1] = r[4]-r[3*id+1], vecA[2] = r[5]-r[3*id+2];
			vecB[0]	= r[3]-rTemp[0], vecB[1] = r[4]-rTemp[1],  vecB[2] = r[5]-rTemp[2];    	
			crossProduct(vecA, vecB, crossP);
			dist 	= norm3d(crossP[0],crossP[1],crossP[2])*rnorm3d(rTemp[0]-r[3*id], rTemp[1]-r[3*id+1], rTemp[2]-r[3*id+2]);
    	}

    	/*else if (t > 1 || t < 0) {
    		// if t is not in the range, it does not lie within the line segment
    		// the equation |r-r[1]|
    		d1   = norm3d(r[3*id]-r[3], r[3*id+1]-r[4], r[3*id+2]-r[5]);

    		// the equation |rTemp-r[1]|
        	d2   = norm3d(rTemp[0]-r[3], rTemp[1]-r[4], rTemp[2]-r[5]);

			dist = fmin(d1, d2); 
    	}*/

		if (dist < rSatellites[0] + rSatellites[1])
			status[id] = 4;
	}
}

// Find distance
__global__ void calcDist(double *r, double *dist) {
	size_t id = blockIdx.x * blockDim.x + threadIdx.x;
	dist[id]  = norm3d(r[3*id], r[3*id+1], r[3*id+2]);
}

// Find eccentricity of all particles
__global__ void calcEccentricity(double *r, double *v, double *m, double *ecc, int numParticles) {
	size_t id = blockIdx.x * blockDim.x + threadIdx.x + 1;
	double L[3];                                                            // angular momentum
	double eccTemp[3];                                                      // hold components of eccentricity vector
	double mu;          					                                // standard gravitational parameter
	double invdist;															// inverse distance between particle and central planet
	
	if (id < numParticles) {
		mu         = m[0] + m[id];	
		invdist    = rnorm3d(r[3*id]-r[0], r[3*id+1]-r[1], r[3*id+2]-r[2]);		
	
		L[0]  	   = (r[3*id+1]-r[1])*v[3*id+2] - (r[3*id+2]-r[2])*v[3*id+1];
		L[1]  	   = (r[3*id+2]-r[2])*v[3*id]   - (r[3*id]-r[0])*v[3*id+2];
		L[2]  	   = (r[3*id]-r[0])*v[3*id+1]   - (r[3*id+1]-r[1])*v[3*id];

		eccTemp[0] = (1./mu) * (v[3*id+1]*L[2] - v[3*id+2]*L[1]) - (r[3*id]-r[0])   * invdist;
		eccTemp[1] = (1./mu) * (v[3*id+2]*L[0] - v[3*id]*L[2])   - (r[3*id+1]-r[1]) * invdist;
		eccTemp[2] = (1./mu) * (v[3*id]*L[1]   - v[3*id+1]*L[0]) - (r[3*id+2]-r[2]) * invdist;

		ecc[id]    = norm3d(eccTemp[0], eccTemp[1], eccTemp[2]); // real eccentricity
	}
}

// Reduce last warp (unrolled) in reduction for A2 operator
template <unsigned int blockSize>
__device__ void warpReduce(volatile double* sdata, int tid) {
	// All statements evaluated at compile time
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8)  sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4)  sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2)  sdata[tid] += sdata[tid + 1];
}
// Reduction kernel for A2 operator for particle 0
template <unsigned int blockSize>
__global__ void reduce(double *g_idata, double *g_odata, unsigned int n) {
    extern __shared__ double sdata[];
	unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    sdata[tid] = 0;
    while (i < n) {
     	sdata[tid] += g_idata[i] + g_idata[i+blockSize];
        i += gridSize;
    }
    __syncthreads();
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/*__global__ void reduce(double *v, double *varr, int numParticles, int s) {
    v[s]   = thrust::reduce(thrust::device, &varr[0], &varr[numParticles]);
    v[1+s] = thrust::reduce(thrust::device, &varr[numParticles], &varr[2*numParticles]);
    v[2+s] = thrust::reduce(thrust::device, &varr[2*numParticles], &varr[3*numParticles]);
}*/

// Function to find
// cross product of two vector array.
void crossProduct2(double *vect_A, double *vect_B, double *cross_P) {
    cross_P[0] = vect_A[1] * vect_B[2] - vect_A[2] * vect_B[1];
    cross_P[1] = vect_A[2] * vect_B[0] - vect_A[0] * vect_B[2];
    cross_P[2] = vect_A[0] * vect_B[1] - vect_A[1] * vect_B[0];
}

// used to calculate the total angular momentum of the system
void linMomentum(double* v, double* m, int numParticles, double *P) {
    *P = 0; 		 // angular momentum
    double  plin[3]; // linear momentum

    for (int i = 0; i < numParticles; i++) {
        plin[0] += m[i]*v[3*i], plin[1] += m[i]*v[3*i+1], plin[2] += m[i]*v[3*i+2];
    *P = sqrt(pow(plin[0], 2) + pow(plin[1], 2) + pow(plin[2], 2));
    }
}

void totalMass(double *m, int numParticles, double* M) {
    *M = 0;
    for (int i = 0; i < numParticles; i++)
        *M += m[i];
}

// used to calculate the total angular momentum of the system
void angMomentum(double* r, double* v, double* m, int numParticles, double *L) {
	*L = 0;
    double Ltemp[3];
	double crossP[3]; // store cross product result
    double dirvec[3]; // distance from planet
    double	p[3]; 	  // linear momentum

    for (int i = 1; i < numParticles; i++) {
        dirvec[0] = -r[0]+r[3*i], dirvec[1] = -r[1]+r[3*i+1], dirvec[2] = -r[2]+r[3*i+2];
             p[0] = m[i]*v[3*i],      p[1] = m[i]*v[3*i+1],	 p[2] = m[i]*v[3*i+2];
        crossProduct2(dirvec, p, crossP);
    	Ltemp[0] += crossP[0], Ltemp[1] += crossP[1], Ltemp[2] += crossP[2];
	}
	*L = sqrt(pow(Ltemp[0], 2) + pow(Ltemp[1], 2) + pow(Ltemp[2], 2));
}

double energynew(double* r, double* v, double* m, int numParticles, double eps) {
    double T = 0;  // kinetic energy
    double U = 0;  // potential energy

    // to hold the vector that points between particle i and particle j
    double* dirvec = (double*)malloc(3 * sizeof(double));

    for (int i = 0; i < numParticles; i++) {
     	T += 0.5 * m[i] * (pow(v[3*i], 2) + pow(v[3*i+1], 2) + pow(v[3*i+2], 2));

        if (i > 0) {
            for (int k = 0; k < 3; k++)
                dirvec[k] = r[k] - r[3*i+k];
            U -= m[0] * m[i] / sqrt(pow(dirvec[0], 2) + pow(dirvec[1], 2) + pow(dirvec[2], 2));
        }
		if (i > 1) {
            for (int k = 0; k < 3; k++)
                dirvec[k] = r[3+k] - r[3*i+k];
            U -= m[1] * m[i] / sqrt(pow(dirvec[0], 2) + pow(dirvec[1], 2) + pow(dirvec[2], 2) + eps*eps);
        }
    }
    free(dirvec);

    return T + U;
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

// Perform the simulation
extern "C" {
void runSim(double *r_h, double *v_h, double *m_h, double dt, int numParticles, int n, double eps, int numSteps, double *ecc_h, double *status_h, double *rSatellites_h, double *dist_h) {
	// Declare useful variables
    size_t i, j; 
	const unsigned int warpSize = 32;
	size_t N                      = 3 * numParticles;
    size_t N_bytes                = N * sizeof(double);
	double rH 					  = 5.37e10/8.8605e9; // scaled 
	double L;	double P;   double M;   double K;
	double L0;  double P0;  double M0;  double K0;
	double semMjrAxis;

	// Make sure the number of particles is multiple of twice the warp size (2*32)
	// for efficiency and reduction
    if (numParticles % (warpSize) != 0) {
    	printf("Error: The number of particles must be a multiple of the warp size (32).\n");
        return;
    }

	// Allocate arrays on device
    double *r_d, *v_d, *m_d, *ecc_d, *varr_d, *rSatellites_d, *status_d, *vTemp_d, *dist_d;
	hipMalloc((void**) &r_d, N_bytes);
    hipMalloc((void**) &v_d, N_bytes);
    hipMalloc((void**) &m_d, N_bytes/3);
	hipMalloc((void**) &varr_d, N_bytes);
	hipMalloc((void**) &status_d, N_bytes/3);
	hipMalloc((void**) &ecc_d, N_bytes/3);
	hipMalloc((void**) &rSatellites_d, 2*sizeof(double));
	hipMalloc((void**) &vTemp_d, numParticles/512*sizeof(double));
	hipMalloc((void**) &dist_d, N_bytes/3);

	// Copy arrays from host to device
    hipMemcpy(r_d, r_h, N_bytes, hipMemcpyHostToDevice);
    hipMemcpy(v_d, v_h, N_bytes, hipMemcpyHostToDevice);
    hipMemcpy(m_d, m_h, N_bytes/3, hipMemcpyHostToDevice);
	hipMemcpy(status_d, status_h, N_bytes/3, hipMemcpyHostToDevice);
	hipMemcpy(rSatellites_d, rSatellites_h, 2*sizeof(double), hipMemcpyHostToDevice);

    //for (i = 0; i < numSteps; i++) {
        // One time step
    /*for (j = 0; j < n; j++) {
        collision<<<numParticles/64, 64>>>(r_d, v_d, status_d, rSatellites_d, numParticles, dt);
        consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
        statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);
        A1_kernel<<<N/512, 512>>>(r_d, v_d, dt/(4*n));
		mergeEject<<<numParticles/64, 64>>>(r_d, status_d, numParticles, rH);
		consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
		statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);

    	A2_kernel<<<numParticles/64, 64>>>(r_d, v_d, m_d, dt/(2*n), varr_d, status_d, numParticles);
	  	reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d, &v_d[0], numParticles);	
		reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+numParticles, &v_d[1], numParticles);
		reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+2*numParticles, &v_d[2], numParticles);

        collision<<<numParticles/64, 64>>>(r_d, v_d, status_d, rSatellites_d, numParticles, dt);
        consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
        statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);
        A1_kernel<<<N/512, 512>>>(r_d, v_d, dt/(4*n));
        mergeEject<<<numParticles/64, 64>>>(r_d, status_d, numParticles, rH);
        consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
        statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);
    }
    B_kernel<<<numParticles/64, 64>>>(r_d, v_d, m_d, varr_d, dt, numParticles, status_d, eps);
    reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d, &v_d[3], numParticles);
    reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+numParticles, &v_d[4], numParticles);
    reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+2*numParticles, &v_d[5], numParticles);

    for (j = 0; j < n; j++) {
    	collision<<<numParticles/64, 64>>>(r_d, v_d, status_d, rSatellites_d, numParticles, dt);
    	consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
    	statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);
    	A1_kernel<<<N/512, 512>>>(r_d, v_d, dt/(4*n));
    	mergeEject<<<numParticles/64, 64>>>(r_d, status_d, numParticles, rH);
   		consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
    	statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);

  		A2_kernel<<<numParticles/64, 64>>>(r_d, v_d, m_d, dt/(2*n), varr_d, status_d, numParticles);
  		reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d, &v_d[0], numParticles);
 		reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+numParticles, &v_d[1], numParticles);
   		reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+2*numParticles, &v_d[2], numParticles);

        collision<<<numParticles/64, 64>>>(r_d, v_d, status_d, rSatellites_d, numParticles, dt);
        consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
        statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);
        A1_kernel<<<N/512, 512>>>(r_d, v_d, dt/(4*n));
        mergeEject<<<numParticles/64, 64>>>(r_d, status_d, numParticles, rH);
        consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
        statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);
    }*/
//}

   /*for (i = 0; i < numParticles; i++)
		printf("%f\n", status_h[i]);

    angMomentum(r_h, v_h, m_h, numParticles, &L0);
    linMomentum(v_h, m_h, numParticles, &P0);
    totalMass(m_h, numParticles, &M0);
    K0 = energynew(r_h, v_h, m_h, numParticles, eps);*/

	/*calcEccentricity<<<numParticles/64, 64>>>(r_d, v_d, m_d, ecc_d, numParticles);
	hipMemcpy(ecc_h, ecc_d, N_bytes/3, hipMemcpyDeviceToHost);

	calcDist<<<numParticles/64, 64>>>(r_d, dist_d);
	hipMemcpy(dist_h, dist_d, N_bytes/3, hipMemcpyDeviceToHost);*/

	/*for (i = 0; i < numSteps; i++) {
        // One time step
    	for (j = 0; j < n; j++) {
        	collision<<<numParticles/64, 64>>>(r_d, v_d, status_d, rSatellites_d, numParticles, dt);
        	consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
        	statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);
        	A1_kernel<<<N/512, 512>>>(r_d, v_d, dt/(4*n));
        	mergeEject<<<numParticles/64, 64>>>(r_d, status_d, numParticles, rH);
        	consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
        	statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);

        	A2_kernel<<<numParticles/64, 64>>>(r_d, v_d, m_d, dt/(2*n), varr_d, status_d, numParticles);
        	reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d, &v_d[0], numParticles);
        	reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+numParticles, &v_d[1], numParticles);
        	reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+2*numParticles, &v_d[2], numParticles);

        	collision<<<numParticles/64, 64>>>(r_d, v_d, status_d, rSatellites_d, numParticles, dt);
        	consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
        	statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);
        	A1_kernel<<<N/512, 512>>>(r_d, v_d, dt/(4*n));
        	mergeEject<<<numParticles/64, 64>>>(r_d, status_d, numParticles, rH);
        	consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
        	statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);
    	}
    	B_kernel<<<numParticles/64, 64>>>(r_d, v_d, m_d, varr_d, dt, numParticles, status_d, eps);
    	reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d, &v_d[3], numParticles);
    	reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+numParticles, &v_d[4], numParticles);
    	reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+2*numParticles, &v_d[5], numParticles);

    	for (j = 0; j < n; j++) {
        	collision<<<numParticles/64, 64>>>(r_d, v_d, status_d, rSatellites_d, numParticles, dt);
        	consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
        	statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);
        	A1_kernel<<<N/512, 512>>>(r_d, v_d, dt/(4*n));
        	mergeEject<<<numParticles/64, 64>>>(r_d, status_d, numParticles, rH);
        	consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
        	statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);

        	A2_kernel<<<numParticles/64, 64>>>(r_d, v_d, m_d, dt/(2*n), varr_d, status_d, numParticles);
        	reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d, &v_d[0], numParticles);
        	reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+numParticles, &v_d[1], numParticles);
        	reduce<512><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+2*numParticles, &v_d[2], numParticles);

        	collision<<<numParticles/64, 64>>>(r_d, v_d, status_d, rSatellites_d, numParticles, dt);
        	consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
        	statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);
        	A1_kernel<<<N/512, 512>>>(r_d, v_d, dt/(4*n));
        	mergeEject<<<numParticles/64, 64>>>(r_d, status_d, numParticles, rH);
        	consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, rSatellites_d);
        	statusUpdate<<<N/64, 64>>>(r_d, v_d, m_d, status_d, numParticles);
    	}
    	//hipMemcpy(r_h, r_d, N_bytes, hipMemcpyDeviceToHost);
    	//hipMemcpy(v_h, v_d, N_bytes, hipMemcpyDeviceToHost);
    	//hipMemcpy(m_h, m_d, N_bytes/3, hipMemcpyDeviceToHost);
	    //hipMemcpy(status_h, status_d, N_bytes/3, hipMemcpyDeviceToHost);
    	//hipMemcpy(rSatellites_h, rSatellites_d, 2*sizeof(double), hipMemcpyDeviceToHost);
		//hipMemcpy(dist_h, dist_d, N_bytes/3, hipMemcpyDeviceToHost);

    	//angMomentum(r_h, v_h, m_h, numParticles, &L);
    	//linMomentum(v_h, m_h, numParticles, &P);
    	//totalMass(m_h, numParticles, &M);
		//K = energynew(r_h, v_h, m_h, numParticles, eps);
		//semMjrAxis = (m_h[0]+m_h[1])*sqrt(r_h[0]*r_h[0]+r_h[1]*r_h[1]+r_h[2]*r_h[2])/(2*(m_h[0]+m_h[1])-sqrt((r_h[0]-r_h[3])*(r_h[0]-r_h[3])+(r_h[1]-r_h[4])*(r_h[1]-r_h[4])+\
		//	(r_h[2]-r_h[5])*(r_h[2]-r_h[5]))*sqrt(v_h[3]*v_h[3]+v_h[4]*v_h[4]+v_h[5]*v_h[5])*sqrt(v_h[3]*v_h[3]+v_h[4]*v_h[4]+v_h[5]*v_h[5]));

		//printf("%.15lf %.15lf %.15lf %.15lf %.15lf %.15lf\n", abs((L-L0)/L0), abs((P-P0)/P0), abs((M-M0)/M0), abs((K-K0)/K0), ecc_h[1], semMjrAxis);
	}*/

	calcEccentricity<<<numParticles/64, 64>>>(r_d, v_d, m_d, ecc_d, numParticles);
    calcDist<<<numParticles/64, 64>>>(r_d, dist_d);
    hipMemcpy(dist_h, dist_d, N_bytes/3, hipMemcpyDeviceToHost);
   	hipMemcpy(ecc_h, ecc_d, N_bytes/3, hipMemcpyDeviceToHost);

	/*for (i = 0; i < numSteps; i++) {
        // One time step
        for (j = 0; j < n; j++) {
            A1_kernel<<<N/512, 512>>>(r_d, v_d, dt/(4*n));

            A2_kernel<<<numParticles/512, 512>>>(r_d, v_d, m_d, dt/(2*n), varr_d, status_d, numParticles);
            reduce<512><<<numParticles/512, 512, 1024*sizeof(double)>>>(varr_d, vTemp_d, numParticles);
			//reduce<SIZE><<<1, numParticles/1024, numParticles/512*sizeof(double)>>>(vTemp_d, &v_d[0], numParticles/512);
            reduce<512><<<numParticles/512, 512, 1024*sizeof(double)>>>(varr_d+numParticles, vTemp_d, numParticles);
            //reduce<SIZE><<<1, numParticles/1024, numParticles/512*sizeof(double)>>>(vTemp_d, &v_d[1], numParticles/512);
			reduce<512><<<numParticles/512, 512, 1024*sizeof(double)>>>(varr_d+2*numParticles, vTemp_d, numParticles);
			//reduce<SIZE><<<1, numParticles/1024, numParticles/512*sizeof(double)>>>(vTemp_d, &v_d[2], numParticles/512);

            A1_kernel<<<N/512, 512>>>(r_d, v_d, dt/(4*n));
        }
		B_kernel<<<numParticles/512, 512>>>(r_d, v_d, m_d, varr_d, dt, numParticles, status_d, eps);
        reduce<512><<<numParticles/512, 512, 1024*sizeof(double)>>>(varr_d, vTemp_d, numParticles);
        //reduce<SIZE><<<1, numParticles/1024, numParticles/512*sizeof(double)>>>(vTemp_d, &v_d[3], numParticles/512);
        reduce<512><<<numParticles/512, 512, 1024*sizeof(double)>>>(varr_d+numParticles, vTemp_d, numParticles);
        //reduce<SIZE><<<1, numParticles/1024, numParticles/512*sizeof(double)>>>(vTemp_d, &v_d[4], numParticles/512);
        reduce<512><<<numParticles/512, 512, 1024*sizeof(double)>>>(varr_d+2*numParticles, vTemp_d, numParticles);
    	//reduce<SIZE><<<1, numParticles/1024, numParticles/512*sizeof(double)>>>(vTemp_d, &v_d[5], numParticles/512);

        for (j = 0; j < n; j++) {
            A1_kernel<<<N/512, 512>>>(r_d, v_d, dt/(4*n));

            A2_kernel<<<numParticles/512, 512>>>(r_d, v_d, m_d, dt/(2*n), varr_d, status_d, numParticles);
            reduce<512><<<numParticles/512, 512, 1024*sizeof(double)>>>(varr_d, vTemp_d, numParticles);
            //reduce<SIZE><<<1, numParticles/1024, numParticles/512*sizeof(double)>>>(vTemp_d, &v_d[0], numParticles/512);
            reduce<512><<<numParticles/512, 512, 1024*sizeof(double)>>>(varr_d+numParticles, vTemp_d, numParticles);
            //reduce<SIZE><<<1, numParticles/1024, numParticles/512*sizeof(double)>>>(vTemp_d, &v_d[1], numParticles/512);
            reduce<512><<<numParticles/512, 512, 1024*sizeof(double)>>>(varr_d+2*numParticles, vTemp_d, numParticles);
    	    //reduce<SIZE><<<1, numParticles/1024, numParticles/512*sizeof(double)>>>(vTemp_d, &v_d[2], numParticles/512);

            A1_kernel<<<N/512, 512>>>(r_d, v_d, dt/(4*n));
		}
	}*/

    /*for (i = 0; i < numSteps; i++) {
        // One time step
        for (j = 0; j < n; j++) {
			collision<<<1, numParticles>>>(r_d, v_d, status_d, rSatellites_d, numParticles, dt);
			consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, 1, rSatellites_d);
            statusUpdate<<<1, N>>>(r_d, v_d, m_d, status_d, numParticles);
            A1_kernel<<<1, N>>>(r_d, v_d, dt/(4*n));
            mergeEject<<<1, numParticles>>>(r_d, status_d, numParticles, rH);
            consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, 0, rSatellites_d);
            statusUpdate<<<1, N>>>(r_d, v_d, m_d, status_d, numParticles);

            A2_kernel<<<1, numParticles>>>(r_d, v_d, m_d, dt/(2*n), varr_d, status_d, numParticles);
            reduce<2><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d, &v_d[0], numParticles);
            reduce<2><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+numParticles, &v_d[1], numParticles);
            reduce<2><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+2*numParticles, &v_d[2], numParticles);

            collision<<<1, numParticles>>>(r_d, v_d, status_d, rSatellites_d, numParticles, dt);
            consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, 1, rSatellites_d);
            statusUpdate<<<1, N>>>(r_d, v_d, m_d, status_d, numParticles);
            A1_kernel<<<1, N>>>(r_d, v_d, dt/(4*n));
            mergeEject<<<1, numParticles>>>(r_d, status_d, numParticles, rH);
            consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, 0, rSatellites_d);
            statusUpdate<<<1, N>>>(r_d, v_d, m_d, status_d, numParticles);
        }
		B_kernel<<<1, numParticles>>>(r_d, v_d, m_d, varr_d, dt, numParticles, status_d, eps);
        reduce<2><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d, &v_d[3], numParticles);
        reduce<2><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+numParticles, &v_d[4], numParticles);
        reduce<2><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+2*numParticles, &v_d[5], numParticles);

        for (j = 0; j < n; j++) {
            collision<<<1, numParticles>>>(r_d, v_d, status_d, rSatellites_d, numParticles, dt);
            consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, 1, rSatellites_d);
            statusUpdate<<<1, N>>>(r_d, v_d, m_d, status_d, numParticles);
            A1_kernel<<<1, N>>>(r_d, v_d, dt/(4*n));
            mergeEject<<<1, numParticles>>>(r_d, status_d, numParticles, rH);
            consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, 0, rSatellites_d);
            statusUpdate<<<1, N>>>(r_d, v_d, m_d, status_d, numParticles);

            A2_kernel<<<1, numParticles>>>(r_d, v_d, m_d, dt/(2*n), varr_d, status_d, numParticles);
            reduce<2><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d, &v_d[0], numParticles);
            reduce<2><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+numParticles, &v_d[1], numParticles);
            reduce<2><<<1, numParticles/2, numParticles*sizeof(double)>>>(varr_d+2*numParticles, &v_d[2], numParticles);

            collision<<<1, numParticles>>>(r_d, v_d, status_d, rSatellites_d, numParticles, dt);
            consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, 1, rSatellites_d);
            statusUpdate<<<1, N>>>(r_d, v_d, m_d, status_d, numParticles);
            A1_kernel<<<1, N>>>(r_d, v_d, dt/(4*n));
            mergeEject<<<1, numParticles>>>(r_d, status_d, numParticles, rH);
            consMomentum<<<1, 1>>>(v_d, m_d, status_d, numParticles, 0, rSatellites_d);
            statusUpdate<<<1, N>>>(r_d, v_d, m_d, status_d, numParticles);
        }
    }*/

    // Copy arrays from device to host
    /*hipMemcpy(r_h, r_d, N_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(v_h, v_d, N_bytes, hipMemcpyDeviceToHost);
	hipMemcpy(m_h, m_d, N_bytes/3, hipMemcpyDeviceToHost);
	hipMemcpy(status_h, status_d, N_bytes/3, hipMemcpyDeviceToHost);
	hipMemcpy(rSatellites_h, rSatellites_d, 2*sizeof(double), hipMemcpyDeviceToHost);

	int h = 0;
	printf("Embryo radius = %.16lf\n", rSatellites_h[0]);
	for (int kk = 0; kk < numParticles; kk++) {
    	if (status_h[kk] == 0) {
        	printf("Index: %d\n", kk);
            printf("New Position\n");
            printf("%.16lf %.16lf %.16lf\n", r_h[3*kk], r_h[3*kk+1], r_h[3*kk+2]);
            printf("New Velocity\n");
            printf("%.16lf %.16lf %.16lf\n", v_h[3*kk], v_h[3*kk+1], v_h[3*kk+2]);
        	h += 1;
		}
    }
	printf("%d\n", h);
	printf("New Mass Planet\n");
	printf("%.16lf\n", m_h[0]);
    printf("New Velocity Planet\n");
    printf("%.16lf %.16lf %.16lf\n", v_h[0], v_h[1], v_h[2]);
	printf("New Mass Embryo\n");
	printf("%.16lf\n", m_h[1]);
   	printf("New Velocity Embryo\n");
    printf("%.16lf %.16lf %.16lf\n", v_h[3], v_h[4], v_h[5]);
	printf("After %d time step(s):\n", numSteps);
    printf("r\n");
    for (i = 0; i < 9; i += 3)
	    printf("%.16lf %.16lf %.16lf\n", r_h[i], r_h[i+1], r_h[i+2]);
    printf("...\n");
    for (i = 3*numParticles - 9; i < 3*numParticles; i += 3)
     	printf("%.16lf %.16lf %.16lf\n", r_h[i], r_h[i+1], r_h[i+2]);
    printf("\n");
    printf("v\n");
    for (i = 0; i < 9; i += 3)
	    printf("%.16lf %.16lf %.16lf\n", v_h[i], v_h[i+1], v_h[i+2]);
    printf("\n");
    printf("...\n");

    for (i = 3*numParticles - 9; i < 3*numParticles; i += 3)
     	printf("%.16lf %.16lf %.16lf\n", v_h[i], v_h[i+1], v_h[i+2]);*/

	// Free allocated memory on host and device
    hipFree(r_d);
    hipFree(v_d);
    hipFree(m_d);
	hipFree(varr_d);
	hipFree(status_d);
    hipFree(ecc_d);
	hipFree(dist_d);
	hipFree(rSatellites_d);
}
}

